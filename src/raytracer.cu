#include "hip/hip_runtime.h"
#include "raytracer.cuh"
#include <float.h>
#include "vec3.cuh"
#include "ray.cuh"
#include "surface.cuh"
#include "surface_list.cuh"
#include "sphere.cuh"
#include "material.cuh"
#include "camera.cuh"

    void Raytracer::check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
        if (result) {
            std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                    file << ":" << line << " '" << func << "' \n";
    
            hipDeviceReset();
            exit(99);
        }
    }
    
    __device__ color ray_trace(const Ray& ray, Surface** world, int max_depth, hiprandState* local_rand_state) {
        Ray current_ray = ray;
        vec3 current_attenuation = vec3(1.0, 1.0, 1.0);
        for (int i = 0; i < max_depth; i ++) {
            Hit_record rec;
            if ((*world)->hit(current_ray, 0.0001, FLT_MAX, rec)) {
                Ray scattered;
                vec3 attenuation;
                if (rec.material->scatter(current_ray, rec, attenuation, scattered, local_rand_state)) {
                    current_attenuation *= attenuation;
                    current_ray = scattered;
                } else {
                    return vec3(0.0, 0.0, 0.0);
                }
            } else {
                vec3 unit_direction = unit_vector(current_ray.direction());
                auto t = 0.5f * (unit_direction.y() + 1.0f);
                color c = (1.0f - t) * vec3(1.0, 1.0, 1.0) + t * vec3(0.5, 0.7, 1.0);
                return current_attenuation * c;
            }
        }
        return vec3(0.0, 0.0, 0.0);
    }
    
    __global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;
        if ((i >= max_x) || (j >= max_y)) return;
        int pixel_index = j * max_x + i;
    
        hiprand_init(clock64(), pixel_index, 0, &rand_state[pixel_index]);
    }
    
    __global__ void render(vec3 *fb, int max_x, int max_y, int max_depth, int samples_per_pixel, Camera** camera, Surface** world, hiprandState* rand_state) {
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        int j = threadIdx.y + blockIdx.y * blockDim.y;
        if ((i >= max_x) || (j >= max_y)) return;
        int pixel_index = j * max_x + i;
        hiprandState local_rand_state = rand_state[pixel_index];
        color pixel_color(0, 0, 0);
        for (int sample = 0; sample < samples_per_pixel; sample ++) {
            float u = static_cast<float>(i + hiprand_uniform(&local_rand_state)) / static_cast<float>(max_x);
            float v = static_cast<float>(j + hiprand_uniform(&local_rand_state)) / static_cast<float>(max_y);
            Ray ray = (*camera)->get_ray(u, v);
            pixel_color += ray_trace(ray, world, max_depth, &local_rand_state);
        }
        rand_state[pixel_index] = local_rand_state;
        pixel_color /= static_cast<float>(samples_per_pixel);
        pixel_color[0] = sqrt(pixel_color[0]);
        pixel_color[1] = sqrt(pixel_color[1]);
        pixel_color[2] = sqrt(pixel_color[2]);
        fb[pixel_index] = pixel_color;
    }

    __global__ void moveCamera(Camera** camera, float* delta) {
        (*camera)->move(delta);
    }
    
    __global__ void create_scene(Surface** d_list, Surface** d_world, Camera** d_camera) {
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            *(d_list + 0) = new Sphere(vec3(0, 0, -1), 0.5, new Diffuse(vec3(0.1, 0.2, 0.5)));
            *(d_list + 1) = new Sphere(vec3(0, -100.5, -1), 100, new Diffuse(vec3(0.8, 0.8, 0.0)));
            *(d_list + 2) = new Sphere(vec3(1, 0, -1), 0.5, new Metal(vec3(0.8, 0.6, 0.2), 0.0));
            *(d_list + 3) = new Sphere(vec3(-1, 0, -1), 0.5, new Dielectric(1.5));
            *(d_list + 4) = new Sphere(vec3(-1, 0, -1), -0.45, new Dielectric(1.5));
            *d_world = new Surface_list(d_list, 5);
            *d_camera = new Camera(vec3(-2, 0, 1), vec3(0, 0, -1), vec3(0, 1, 0), 30, 2.0, 0.1, 10.0, 0.0, 1.0);
        }
    }
    
    __global__ void free_scene(Surface** d_list, Surface** d_world, Camera** d_camera) {
        for (int i = 0; i < 5; i ++) {
            delete ((Sphere*)d_list[i])->material;
            delete d_list[i];
        }
        delete *(d_world);
        delete *(d_camera);
    }

    Raytracer::Raytracer(int SCREENWIDTH, int SCREENHEIGHT) : SCREENWIDTH(SCREENWIDTH), SCREENHEIGHT(SCREENHEIGHT){
        // GPU settings
        int num_pixels = SCREENWIDTH * SCREENHEIGHT;
        size_t fb_size = num_pixels * sizeof(vec3);   // (r, g, b)

        // camera
        checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera*)));
        checkCudaErrors(hipMalloc((void**)&camera_displacement, 5 * sizeof(float)));

        // allocate framebuffer
        checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

        // allocate random state
        checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels * sizeof(hiprandState)));

        // create scene
        checkCudaErrors(hipMalloc((void**)&d_list, 2*sizeof(Surface*)));
        checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Surface*)));
        create_scene<<<1, 1>>>(d_list, d_world, d_camera);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
    }

    Raytracer::~Raytracer() {
        checkCudaErrors(hipDeviceSynchronize());
        free_scene<<<1, 1>>>(d_list, d_world, d_camera);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(camera_displacement));
        checkCudaErrors(hipFree(d_list));
        checkCudaErrors(hipFree(d_world));
        checkCudaErrors(hipFree(fb));
    
        hipDeviceReset();
    }

    void Raytracer::update(std::vector<std::vector<std::vector<int>>> &buffer, float* delta) {
        hipMemcpy(camera_displacement, delta, 5 * sizeof(float), hipMemcpyHostToDevice);
        moveCamera<<<1, 1>>>(d_camera, camera_displacement);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        dim3 blocks(SCREENWIDTH / block_width + 1, SCREENHEIGHT / block_height + 1);
        dim3 threads(block_width, block_height);
        render_init<<<blocks, threads>>>(SCREENWIDTH, SCREENHEIGHT, d_rand_state);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        render<<<blocks, threads>>>(fb, SCREENWIDTH, SCREENHEIGHT, MAX_DEPTH, SAMPLES_PER_PIXEL, d_camera, d_world, d_rand_state);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
        
        for (int j = SCREENHEIGHT - 1; j >= 0; j --) {
            for (int i = 0; i < SCREENWIDTH; i ++) {
                size_t pixel_index = j * SCREENWIDTH + i;
                auto r = fb[pixel_index].r();
                auto g = fb[pixel_index].g();
                auto b = fb[pixel_index].b();
                int ir = static_cast<int>(255.999 * r);
                int ig = static_cast<int>(255.999 * g);
                int ib = static_cast<int>(255.999 * b);
                buffer[SCREENHEIGHT - 1 - j][i][0] = ir;
                buffer[SCREENHEIGHT - 1 - j][i][1] = ig;
                buffer[SCREENHEIGHT - 1 - j][i][2] = ib;
            }
        }
    }